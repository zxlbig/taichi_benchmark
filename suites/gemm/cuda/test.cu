#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <sys/time.h>
#include "utils.cuh"
#define MYSGEMM mysgemm_naive // select the kernel here

int main(int argc, char **argv){
    if (argc < 2) {
        printf("Please select a kernel (range 0 - 11, here 0 is for NVIDIA cuBLAS).\n");
        printf("Please select a kernel size -1 for automated.\n");
        exit(-1);
    }
    int SIZE[24];
    for (int i=0;i<24;i++) SIZE[i]=(i+1)<<8;
    int kernel_num=atoi(argv[1]);
    int kernel_size= atoi(argv[2]);
    if (kernel_num<0||kernel_num>11) {
        printf("Please enter a valid kernel number (0-11).\n");
        exit(-2);
    }
    int m, n, k,max_size;
    int n_count,N=100,upper_limit;
    // if (kernel_num<=4&&kernel_num!=0) upper_limit=8;
    // else 
    upper_limit=(sizeof(SIZE)/sizeof(int));
    max_size=SIZE[upper_limit-1];
    FLOAT *A=NULL,*B=NULL,*C=NULL,*C_ref=NULL;//host matrices
    FLOAT *dA=NULL,*dB=NULL,*dC=NULL,*dC_ref=NULL;//device matrices
    FLOAT alpha = 1.0, beta = 0.;//two arbitary input parameters
    float elapsed_time;
    hipblasHandle_t err; hipblasCreate(&err);
    hipEvent_t beg, end;
    hipEventCreate(&beg);
    hipEventCreate(&end);
    A=(FLOAT *)malloc(sizeof(FLOAT)*max_size*max_size);
    B=(FLOAT *)malloc(sizeof(FLOAT)*max_size*max_size);
    C=(FLOAT *)malloc(sizeof(FLOAT)*max_size*max_size);
    C_ref=(FLOAT *)malloc(sizeof(FLOAT)*max_size*max_size);
    randomize_matrix(A,max_size*max_size);randomize_matrix(B,max_size*max_size);
    randomize_matrix(C,max_size*max_size);copy_matrix(C,C_ref,max_size*max_size);
    CUDA_CALLER(hipMalloc((void**) &dA, sizeof(FLOAT)*max_size*max_size));
    CUDA_CALLER(hipMalloc((void**) &dB, sizeof(FLOAT)*max_size*max_size));
    CUDA_CALLER(hipMalloc((void**) &dC, sizeof(FLOAT)*max_size*max_size));
    CUDA_CALLER(hipMalloc((void**) &dC_ref, sizeof(FLOAT)*max_size*max_size));
    CUDA_CALLER(hipMemcpy(dA, A, sizeof(FLOAT)*max_size*max_size, hipMemcpyHostToDevice));
    CUDA_CALLER(hipMemcpy(dB, B, sizeof(FLOAT)*max_size*max_size, hipMemcpyHostToDevice));
    CUDA_CALLER(hipMemcpy(dC, C, sizeof(FLOAT)*max_size*max_size, hipMemcpyHostToDevice));
    CUDA_CALLER(hipMemcpy(dC_ref, C_ref, sizeof(FLOAT)*max_size*max_size, hipMemcpyHostToDevice));
    if (kernel_size != -1) {
        upper_limit = 1;
    }
    for (int i_count=0;i_count<upper_limit;i_count++){
         m=n=k=kernel_size;
        printf("\nM=N=K=%d:\n",m);
        if (kernel_num != 0){//not cuBLAS
            hipblasSgemm(err, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, dA, m, dB, k, &beta, dC_ref, m);
            test_kernel(kernel_num,m,n,k,alpha,dA,dB,beta,dC);
            hipDeviceSynchronize();
            hipMemcpy(C, dC, sizeof(FLOAT)*m*n, hipMemcpyDeviceToHost);
            hipMemcpy(C_ref, dC_ref, sizeof(FLOAT)*m*n, hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            // if (!verify_matrix(C_ref,C,m*n)) {
            //     printf("Failed to pass the correctness verification against NVIDIA cuBLAS. Exited.\n");
            //     exit(-3);
            // }
        }


        hipEventRecord(beg);
        if (kernel_num != 0){
            for (n_count=0;n_count<N;n_count++){
                test_kernel(kernel_num,m,n,k,alpha,dA,dB,beta,dC);
            }
        }else{
            for (n_count=0;n_count<N;n_count++){
                test_kernel(kernel_num,m,n,k,alpha,dA,dB,beta,dC, err);
            }
        }

        hipEventRecord(end);
        hipEventSynchronize(beg);
        hipEventSynchronize(end);
        hipEventElapsedTime(&elapsed_time, beg, end);
        elapsed_time /= 1000.;

        printf("Average elasped time: %f second, performance: %f GFLOPS.\n", elapsed_time/N,2.*1e-9*N*m*n*k/elapsed_time);
        fflush(stdout);
        copy_matrix(C_ref,C,m*n);//sync C with cuBLAS to prepare for the next run
    }
    hipDeviceSynchronize();
    free(A);free(B);free(C);free(C_ref);
    hipFree(dA);hipFree(dB);hipFree(dC);hipFree(dC_ref);
    hipDeviceSynchronize();
    return 0;
}
